#include "hip/hip_runtime.h"
#include "BioTK/hip/hip_runtime.h"

__global__ void biotk_cuda_dot_gpu( double *a, double *b, double *c) {
    __shared__ double temp[CUDA_BLOCK_SIZE];
    temp[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];

    __syncthreads();

    if( 0 == threadIdx.x ) {
        double sum = 0;
        for( int i = CUDA_BLOCK_SIZE-1; i >= 0; i-- ){
            sum += temp[i];
        }
        *c = sum;
    }
}

double biotk_cuda_dot(const double* a, const double* b) {
    int size = CUDA_BLOCK_SIZE * sizeof( double ); 

    double *dev_a, *dev_b, *dev_o; // device copies of a, b, c
    double *o = (double *)malloc( sizeof( double ) );

    hipMalloc( (void**)&dev_a, size );
    hipMalloc( (void**)&dev_b, size );
    hipMalloc( (void**)&dev_o, sizeof( double ) );

    hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );

    // launch dot() kernel with 1 block and N threads
    biotk_cuda_dot_gpu<<< 1, CUDA_BLOCK_SIZE >>>( dev_a, dev_b, dev_o);
    // copy device result back to host copy of c
    hipMemcpy( o, dev_o, sizeof( double ), hipMemcpyDeviceToHost );

    double result = *o;
    free( o );
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_o );
    return result;
}
